#include "hip/hip_runtime.h"
#include <chrono>
#include <iomanip>
#include <iostream>
#include <random>
#include "Timer.cuh"
#include "CheckError.cuh"
using namespace timer;

const int BLOCK_SIZE_X = 16;
const int BLOCK_SIZE_Y = 16;

__global__
void matrixMultiplicationKernel(const int* d_matrixA,
                                const int* d_matrixB,
                                int        N,
                                int*       d_matrixC) {
    // Calculate the row index of the Pd element and M
    int Row = blockIdx.y*blockDim.y + threadIdx.y;
    // Calculate the column index of Pd and N
    int Col = blockIdx.x*blockDim.x + threadIdx.x;
    
    float Pvalue = 0;
    // each thread computes one element of the block sub-matrix
    for (int k = 0; k < N; ++k)
        Pvalue += d_matrixA[Row*N+k] * d_matrixB[Col+k*N];
    
    d_matrixC[Row*N+Col] = Pvalue;
}

const int N = 1024;

int main() {
    Timer<DEVICE> TM_device;
    Timer<HOST>   TM_host;
    // -------------------------------------------------------------------------
    // HOST MEMORY ALLOCATION
    int* h_matrixA    = new int[N * N];
    int* h_matrixB    = new int[N * N];
    int* h_matrix_tmp = new int[N * N]; // <-- used for device result
    int* h_matrixC    = new int[N * N];

    // -------------------------------------------------------------------------
    // HOST INITILIZATION
    unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::default_random_engine generator(seed);
    std::uniform_int_distribution<int> distribution(1, 100);

    for (int i = 0; i < N * N; i++) {
        h_matrixA[i] = distribution(generator);
        h_matrixB[i] = distribution(generator);
    }
    // -------------------------------------------------------------------------
    // HOST EXECUTIION
    std::cout<<"Starting computation on HOST.."<<std::endl;
    TM_host.start();

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            int sum = 0;
            for (int k = 0; k < N; k++)
                 sum += h_matrixA[i * N + k] * h_matrixB[k * N + j];
            h_matrixC[i * N + j] = sum;
        }
    }

    TM_host.stop();
    TM_host.print("MatrixMultiplication host:   ");

    // -------------------------------------------------------------------------
    // DEVICE MEMORY ALLOCATION
    int *d_matrixA, *d_matrixB, *d_matrixC;
    SAFE_CALL( hipMalloc( &d_matrixA, N*N * sizeof(int) ));
    SAFE_CALL( hipMalloc( &d_matrixB, N*N * sizeof(int) ));
    SAFE_CALL( hipMalloc( &d_matrixC, N*N * sizeof(int) ));

    // -------------------------------------------------------------------------
    // COPY DATA FROM HOST TO DEVIE
    SAFE_CALL( hipMemcpy( d_matrixA, h_matrixA, N*N * sizeof(int), hipMemcpyHostToDevice));
    SAFE_CALL( hipMemcpy( d_matrixB, h_matrixB, N*N * sizeof(int), hipMemcpyHostToDevice));

    // -------------------------------------------------------------------------
    // DEVICE INIT
    dim3 DimGrid(N/BLOCK_SIZE_X, N/BLOCK_SIZE_Y, 1);
    if (N%BLOCK_SIZE_X) DimGrid.x++;
    if (N%BLOCK_SIZE_Y) DimGrid.y++;
    dim3 DimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y, 1);
    
    // -------------------------------------------------------------------------
    // DEVICE EXECUTION
    TM_device.start();
    matrixMultiplicationKernel<<< DimGrid,DimBlock>>> (d_matrixA, d_matrixB, N, d_matrixC);

    CHECK_CUDA_ERROR
    TM_device.stop();
    TM_device.print("MatrixMultiplication device: ");

    std::cout << std::setprecision(1)
              << "Speedup: " << TM_host.duration() / TM_device.duration()
              << "x\n\n";

    // -------------------------------------------------------------------------
    // COPY DATA FROM DEVICE TO HOST
    SAFE_CALL( hipMemcpy( h_matrix_tmp, d_matrixC, N*N * sizeof(int), hipMemcpyDeviceToHost));

    // -------------------------------------------------------------------------
    // RESULT CHECK
    for (int i = 0; i < N * N; i++) {
        if (h_matrixC[i] != h_matrix_tmp[i]) {
            std::cerr << "wrong result at: ("
                      << (i / N) << ", " << (i % N) << ")"
                      << "\nhost:   " << h_matrixC[i]
                      << "\ndevice: " << h_matrix_tmp[i] << "\n\n";
            hipDeviceReset();
            std::exit(EXIT_FAILURE);
        }
    }
    std::cout << "<> Correct\n\n";

    // -------------------------------------------------------------------------
    // HOST MEMORY DEALLOCATION
    delete[] h_matrixA;
    delete[] h_matrixB;
    delete[] h_matrixC;
    delete[] h_matrix_tmp;

    // -------------------------------------------------------------------------
    // DEVICE MEMORY DEALLOCATION
    SAFE_CALL( hipFree( d_matrixA ) );
    SAFE_CALL( hipFree( d_matrixB ) );
    SAFE_CALL( hipFree( d_matrixC ) );

    // -------------------------------------------------------------------------
    hipDeviceReset();
}
