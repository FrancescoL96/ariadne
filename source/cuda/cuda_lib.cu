#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <math.h>

#include "./cuda_lib.hpp"
#include "CheckError.cuh"

#define ADD 0
#define SUB 1
#define MUL 2
#define DIV 3

#define round_up 0
#define round_down 1
#define round_to_nearest 2
#define round_toward_zero 3

const int BLOCK_SIZE_X = 16;
const int BLOCK_SIZE_Y = 16;

__global__
void matrixMultiplicationKernel(const int* d_matrixA,
                                const int* d_matrixB,
                                int        N,
                                int*       d_matrixC) {
    int Row = blockIdx.y*blockDim.y + threadIdx.y;
    int Col = blockIdx.x*blockDim.x + threadIdx.x;
    
    int Pvalue = 0;
    if (Row < N && Col < N) {
        for (int k = 0; k < N; ++k)
            Pvalue += d_matrixA[Row*N+k] * d_matrixB[Col+k*N];        

        d_matrixC[Row*N+Col] = Pvalue;
    }
}

__global__
void cuda_operation_float_ru (float first_value, float second_value, int operation, float * res) {
    switch (operation) {
        case ADD:
            * res = __fadd_ru(first_value, second_value);
            break;
        case SUB:
            * res = __fadd_ru(first_value, -second_value);
            break;
        case MUL:
            * res = __fmul_ru (first_value, second_value);
            break;
        case DIV:
            * res = __fdiv_ru (first_value, second_value);
            break;
    }
}

__global__
void cuda_operation_float_rd (float first_value, float second_value, int operation, float * res) {
    switch (operation) {
        case ADD:
            * res = __fadd_rd(first_value, second_value);
            break;
        case SUB:
            * res = __fadd_rd(first_value, -second_value);
            break;
        case MUL:
            * res = __fmul_rd(first_value, second_value);
            break;
        case DIV:
            * res = __fdiv_rd(first_value, second_value);
            break;
    }
}

__global__
void cuda_operation_double_ru(double first_value, double second_value, int operation, double * res) {
    switch (operation) {
        case ADD:
            * res = __dadd_ru(first_value, second_value);
            break;
        case SUB:
            * res = __dadd_ru(first_value, -second_value);
            break;
        case MUL:
            * res = __dmul_ru(first_value, second_value);
            break;
        case DIV:
            * res = __ddiv_ru(first_value, second_value);
            break;
    }
}

__global__
void cuda_operation_double_rd(double first_value, double second_value, int operation, double * res) {
    switch (operation) {
        case ADD:
            * res = __dadd_rd(first_value, second_value);
            break;
        case SUB:
            * res = __dadd_rd(first_value, -second_value);
            break;
        case MUL:
            * res = __dmul_rd(first_value, second_value);
            break;
        case DIV:
            * res = __ddiv_rd(first_value, second_value);
            break;
    }
}

void ariadne_cuda::function(const int N, int * h_matrixA, int * h_matrixB, int * h_matrixC) {
    int *d_matrixA, *d_matrixB, *d_matrixC;
    SAFE_CALL(hipMalloc( &d_matrixA, N*N * sizeof(int) ));
    SAFE_CALL(hipMalloc( &d_matrixB, N*N * sizeof(int) ));
    SAFE_CALL(hipMalloc( &d_matrixC, N*N * sizeof(int) ));

    SAFE_CALL(hipMemcpy( d_matrixA, h_matrixA, N*N * sizeof(int), hipMemcpyHostToDevice));
    SAFE_CALL(hipMemcpy( d_matrixB, h_matrixB, N*N * sizeof(int), hipMemcpyHostToDevice));

    dim3 DimGrid(N/BLOCK_SIZE_X, N/BLOCK_SIZE_Y, 1);
    if (N%BLOCK_SIZE_X) DimGrid.x++;
    if (N%BLOCK_SIZE_Y) DimGrid.y++;
    dim3 DimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y, 1);
    
    matrixMultiplicationKernel<<< DimGrid,DimBlock>>> (d_matrixA, d_matrixB, N, d_matrixC);
    CHECK_CUDA_ERROR

    SAFE_CALL(hipMemcpy( h_matrixC, d_matrixC, N*N * sizeof(int), hipMemcpyDeviceToHost));

    std::cout << "GPU: " << std::endl;
    for (int i = 0; i < N * N; i++){
        if (i % N == 0){
            std::cout << std::endl;
        }
        std::cout << h_matrixC[i] << " ";
    }
    std::cout << std::endl;

    SAFE_CALL(hipFree(d_matrixA));
    SAFE_CALL(hipFree(d_matrixB));
    SAFE_CALL(hipFree(d_matrixC));
}

float ariadne_cuda::float_approximation (float first_value, float second_value, int operation, int rounding) {
    float * res_d;
    float * res_h = new float();

    SAFE_CALL(hipMalloc(&res_d, sizeof(float)));
    switch (rounding) {
        case round_down:
            cuda_operation_float_rd <<< 1, 1 >>> (first_value, second_value, operation, res_d);
            break;
        case round_up:
            cuda_operation_float_ru <<< 1, 1 >>> (first_value, second_value, operation, res_d);
            break;
        case round_to_nearest:
            
            break;
        case round_toward_zero:
            
            break;
    }
    CHECK_CUDA_ERROR

    SAFE_CALL(hipMemcpy(res_h, res_d, sizeof(float), hipMemcpyDeviceToHost));
    SAFE_CALL(hipFree(res_d));
    
    return * res_h;
}

double ariadne_cuda::double_approximation (double first_value, double second_value, int operation, int rounding) {
    double * res_d;
    double * res_h = new double();

    SAFE_CALL(hipMalloc(&res_d, sizeof(double)));
    switch (rounding) {
        case round_down:
            cuda_operation_double_rd <<< 1, 1 >>> (first_value, second_value, operation, res_d);
            break;
        case round_up:
            cuda_operation_double_ru <<< 1, 1 >>> (first_value, second_value, operation, res_d);
            break;
        case round_to_nearest:
            
            break;
        case round_toward_zero:
            
            break;
    }
    CHECK_CUDA_ERROR
    SAFE_CALL(hipMemcpy(res_h, res_d, sizeof(double), hipMemcpyDeviceToHost));
    SAFE_CALL(hipFree(res_d));

    return * res_h;
}

double * ariadne_cuda::mallocManagedDouble(int size) {
    double * var;
    SAFE_CALL(hipMallocManaged(&var, size * sizeof(double)));
    for (int i = 0; i < size; i++) {
        var[i] = double(0);
    }
    return var;
}

int * ariadne_cuda::mallocManagedInt(int size) {
    int * var;
    SAFE_CALL(hipMallocManaged(&var, size * sizeof(int)));
    for (int i = 0; i < size; i++) {
        var[i] = int(0);
    }
    return var;
}

__global__
void sum_index (int * x_index_vector, int * y_index_matrix, int ya_len, int y_size) {
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    y_index_matrix[row * y_size + col] += x_index_vector[col];
}

/* Note:
 * This kernel is not implemented in the most efficient way possible, local variables should be omitted
 */
__global__
void mul_value (double x_value, double x_value_neg, double * y_value_vector, int y_size, double * error) {
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    double u = __dmul_ru(y_value_vector[col], x_value);
    double ml = __dmul_ru(y_value_vector[col], x_value_neg);
    double add = __dadd_ru(u, ml);
    double two = 2.0;
    error[col] = __ddiv_ru(add, two);
    y_value_vector[col] = __dmul_rn(y_value_vector[col], x_value);
}

void ariadne_cuda::_ifma(int *x_index_vector, double x_value, double x_value_neg, 
    int *y_index_matrix, double *y_value_vector, int ya_len, int y_size, double * error)
{
    sum_index <<< ya_len, y_size >>> (x_index_vector, y_index_matrix, ya_len, y_size);
    mul_value <<< 1, y_size >>> (x_value, x_value_neg, y_value_vector, y_size, error);
    CHECK_CUDA_ERROR
}