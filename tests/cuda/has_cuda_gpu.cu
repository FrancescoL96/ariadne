#include <iostream>
#include <hip/hip_runtime.h>

#include "../../source/cuda/CheckError.cuh"

int main() {
    int device_count, device;
    int gpu_device_count = 0;
    struct hipDeviceProp_t properties;
    SAFE_CALL(hipGetDeviceCount(&device_count));
    
    for (device = 0; device < device_count; ++device) {
        hipGetDeviceProperties(&properties, device);
        if (properties.major != 9999)
            gpu_device_count++;
    }
    printf("%d GPU CUDA device(s) found", gpu_device_count);

    if (gpu_device_count > 0)
        return 0;
    else
        return 1;
}